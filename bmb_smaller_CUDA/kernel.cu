#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <stdio.h>
#include <Windows.h>
#include <time.h>
#include <thread>
#include <omp.h>
#include <vector>
#include <math.h>
#include <cmath>

using namespace std;

struct img_param {
	int size;
	int pixels_adress;
	int width;
	int height;
	short int bits_per_pixel;
};
struct px_arr {
	unsigned int *r;
	unsigned int *g;
	unsigned int *b;
};


px_arr reading(int dimensions1, int dimensions2, ifstream &file) {
	unsigned int r = 0;
	unsigned int g = 0;
	unsigned int b = 0;

	px_arr old;
	old.r = new unsigned int[dimensions1 * dimensions2];
	old.g = new unsigned int[dimensions1 * dimensions2];
	old.b = new unsigned int[dimensions1 * dimensions2];
	
	for (int i = 0; i < dimensions1 * dimensions2; i++) {
		file.read((char*)&b, 1);
		file.read((char*)&g, 1);
		file.read((char*)&r, 1);
		old.r[i] = r;
		old.g[i] = g;
		old.b[i] = b;

	}
	return old;
}

px_arr small_width(px_arr old, int dimensions1, int dimensions2, int new_width) {
	int dop = 4 - dimensions1 % 4;
	if (dop == 4) {
		dop = 0;
	}

	int null = 0;

	px_arr young;
	young.r = new unsigned int[new_width * dimensions2];
	young.g = new unsigned int[new_width * dimensions2];
	young.b = new unsigned int[new_width * dimensions2];


	for (int i = 0; i < dimensions2; i++) {
		for (int j = 0; j < new_width; j++) {
			young.r[new_width * i + j] = old.r[(int)round(((float)dimensions1 / (float)new_width) * j + dimensions1 * i)];
			young.g[new_width * i + j] = old.g[(int)round(((float)dimensions1 / (float)new_width) * j + dimensions1 * i)];
			young.b[new_width * i + j] = old.b[(int)round(((float)dimensions1 / (float)new_width) * j + dimensions1 * i)];
		}
	}



	return young;
}

px_arr small_height(px_arr old, int new_width, int dimensions2, int new_height) {
	px_arr young;
	young.r = new unsigned int[new_height * new_width];
	young.g = new unsigned int[new_height * new_width];
	young.b = new unsigned int[new_height * new_width];

	for (int i = 0; i < new_height; i++) {
		for (int j = 0; j < new_width; j++) {
			young.r[new_width * i + j] = old.r[(int)round(((float)dimensions2 / (float)new_height) * i) * new_width + j];
			young.g[new_width * i + j] = old.g[(int)round(((float)dimensions2 / (float)new_height) * i) * new_width + j];
			young.b[new_width * i + j] = old.b[(int)round(((float)dimensions2 / (float)new_height) * i) * new_width + j];
		}
	}

	return young;
}

void writing(px_arr young, int new_height, ofstream &os, int new_width) {
	for (int i = 0; i < new_height; i++) {
		for (int j = 0; j < new_width; j++) {
			os.write(reinterpret_cast<char*>(&young.b[new_width * i + j]), sizeof(char));
			os.write(reinterpret_cast<char*>(&young.g[new_width * i + j]), sizeof(char));
			os.write(reinterpret_cast<char*>(&young.r[new_width * i + j]), sizeof(char));
		}
	}

}


int main(int argc, char **argv)
{
	setlocale(LC_CTYPE, "rus");
	int size = 0, pixels_adress = 0, width = 0, height = 0;
	short int bits_per_pixel = 0;

	ifstream file("nature.bmp", ios::in | ios::binary);

	// ��������� �� 2 ����
	file.seekg(2, ios::beg);

	// ��������� ������ �����
	file.read((char*)&size, sizeof(int));
	std::cout << "Size: " << size << endl;

	// ��������� �� 10 ����
	file.seekg(10, ios::beg);

	// ��������� �����, ��� ����� ���������� � ��������
	file.read((char*)&pixels_adress, sizeof(int));
	std::cout << "pixels_adress: " << pixels_adress << endl;

	// ��������� �� 18 ����
	file.seekg(18, ios::beg);

	//��������� ������ ��������
	file.read((char*)&width, sizeof(int));
	std::cout << "width: " << width << endl;

	// ��������� �� 22 ����
	file.seekg(22, ios::beg);

	//��������� ������ ��������
	file.read((char*)&height, sizeof(int));
	std::cout << "height: " << height << endl;

	// ��������� �� 28 ����
	file.seekg(28, ios::beg);

	//��������� ���������� ��� �� �������
	file.read((char*)&bits_per_pixel, sizeof(short int));
	std::cout << "bits_per_pixel: " << bits_per_pixel << endl;

	//��������� � ���� ������ ��������
	file.seekg(pixels_adress, ios::beg);

	float new_width, new_height;
	std::cout << "����� ������ ����������� � �������� (������ ������� � ������� �� 4)" << endl;
	std::cin >> new_width;
	std::cout << endl;

	std::cout << "����� ������ ����������� � ��������" << endl;
	std::cin >> new_height;
	std::cout << endl;


	std::ofstream os("temp_0.bmp", std::ios::binary);
	unsigned char signature[2] = { 'B', 'M' };
	unsigned int fileSize = 14 + 40 + new_width * new_height * 3;
	unsigned int reserved = 0;
	unsigned int offset = 14 + 40;
	unsigned int headerSize = 40;
	unsigned int dimensions1 = new_width;
	unsigned int dimensions2 = new_height;
	unsigned short colorPlanes = 1;
	unsigned short bpp = 24;
	unsigned int compression = 0;
	unsigned int imgSize = new_width * new_height * 3;
	unsigned int resolution[2] = { 2795, 2795 };
	unsigned int pltColors = 0;
	unsigned int impColors = 0;
	os.write(reinterpret_cast<char*>(signature), sizeof(signature));
	os.write(reinterpret_cast<char*>(&fileSize), sizeof(fileSize));
	os.write(reinterpret_cast<char*>(&reserved), sizeof(reserved));
	os.write(reinterpret_cast<char*>(&offset), sizeof(offset));
	os.write(reinterpret_cast<char*>(&headerSize), sizeof(headerSize));
	os.write(reinterpret_cast<char*>(&dimensions1), sizeof(dimensions1));
	os.write(reinterpret_cast<char*>(&dimensions2), sizeof(dimensions2));
	os.write(reinterpret_cast<char*>(&colorPlanes), sizeof(colorPlanes));
	os.write(reinterpret_cast<char*>(&bpp), sizeof(bpp));
	os.write(reinterpret_cast<char*>(&compression), sizeof(compression));
	os.write(reinterpret_cast<char*>(&imgSize), sizeof(imgSize));
	os.write(reinterpret_cast<char*>(resolution), sizeof(resolution));
	os.write(reinterpret_cast<char*>(&pltColors), sizeof(pltColors));
	os.write(reinterpret_cast<char*>(&impColors), sizeof(impColors));

	px_arr old, young_w, young_wh;
	old = reading(width, height, file);
	young_w = small_width(old, width, height, new_width);
	young_wh = small_height(young_w, new_width, height, new_height);
	writing(young_wh, new_height, os, new_width);

	os.close();

	return 0;
}