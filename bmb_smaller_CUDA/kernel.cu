#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <stdio.h>
#include <Windows.h>
#include <time.h>
#include <thread>
#include <omp.h>
#include <vector>
#include <math.h>
#include <cmath>

using namespace std;

struct img_param {
	int size;
	int pixels_adress;
	int width;
	int height;
	short int bits_per_pixel;
};
struct px_arr {
	unsigned int *r;
	unsigned int *g;
	unsigned int *b;
};


px_arr reading(int dimensions1, int dimensions2, ifstream &file) {
	unsigned int r = 0;
	unsigned int g = 0;
	unsigned int b = 0;

	px_arr old;
	old.r = new unsigned int[dimensions1 * dimensions2];
	old.g = new unsigned int[dimensions1 * dimensions2];
	old.b = new unsigned int[dimensions1 * dimensions2];
	
	for (int i = 0; i < dimensions1 * dimensions2; i++) {
		file.read((char*)&b, 1);
		file.read((char*)&g, 1);
		file.read((char*)&r, 1);
		old.r[i] = r;
		old.g[i] = g;
		old.b[i] = b;

	}
	return old;
}

void small_width(unsigned int *r, unsigned int *g, unsigned int *b, int width, int height, int new_width, unsigned int *yr, unsigned int *yg, unsigned int *yb) {
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < new_width; j++) {
			yr[new_width * i + j] = r[(int)round(((float)width / (float)new_width) * j + width * i)];
			yg[new_width * i + j] = g[(int)round(((float)width / (float)new_width) * j + width * i)];
			yb[new_width * i + j] = b[(int)round(((float)width / (float)new_width) * j + width * i)];
		}
	}
}

void small_height(unsigned int *r, unsigned int *g, unsigned int *b, int new_width, int height, int new_height, unsigned int *yr, unsigned int *yg, unsigned int *yb) {
	for (int i = 0; i < new_height; i++) {
		for (int j = 0; j < new_width; j++) {
			yr[new_width * i + j] = r[(int)round(((float)height / (float)new_height) * i) * new_width + j];
			yg[new_width * i + j] = g[(int)round(((float)height / (float)new_height) * i) * new_width + j];
			yb[new_width * i + j] = b[(int)round(((float)height / (float)new_height) * i) * new_width + j];
		}
	}
}

void writing(px_arr young, int new_height, ofstream &os, int new_width) {
	for (int i = 0; i < new_height; i++) {
		for (int j = 0; j < new_width; j++) {
			os.write(reinterpret_cast<char*>(&young.b[new_width * i + j]), sizeof(char));
			os.write(reinterpret_cast<char*>(&young.g[new_width * i + j]), sizeof(char));
			os.write(reinterpret_cast<char*>(&young.r[new_width * i + j]), sizeof(char));
		}
	}

}


int main(int argc, char **argv)
{
	setlocale(LC_CTYPE, "rus");
	int size = 0, pixels_adress = 0, width = 0, height = 0;
	short int bits_per_pixel = 0;

	ifstream file("nature.bmp", ios::in | ios::binary);

	// ��������� �� 2 ����
	file.seekg(2, ios::beg);

	// ��������� ������ �����
	file.read((char*)&size, sizeof(int));
	std::cout << "Size: " << size << endl;

	// ��������� �� 10 ����
	file.seekg(10, ios::beg);

	// ��������� �����, ��� ����� ���������� � ��������
	file.read((char*)&pixels_adress, sizeof(int));
	std::cout << "pixels_adress: " << pixels_adress << endl;

	// ��������� �� 18 ����
	file.seekg(18, ios::beg);

	//��������� ������ ��������
	file.read((char*)&width, sizeof(int));
	std::cout << "width: " << width << endl;

	// ��������� �� 22 ����
	file.seekg(22, ios::beg);

	//��������� ������ ��������
	file.read((char*)&height, sizeof(int));
	std::cout << "height: " << height << endl;

	// ��������� �� 28 ����
	file.seekg(28, ios::beg);

	//��������� ���������� ��� �� �������
	file.read((char*)&bits_per_pixel, sizeof(short int));
	std::cout << "bits_per_pixel: " << bits_per_pixel << endl;

	//��������� � ���� ������ ��������
	file.seekg(pixels_adress, ios::beg);

	int new_width, new_height;
	std::cout << "����� ������ ����������� � �������� (������ ������� � ������� �� 4)" << endl;
	std::cin >> new_width;
	std::cout << endl;

	std::cout << "����� ������ ����������� � ��������" << endl;
	std::cin >> new_height;
	std::cout << endl;


	std::ofstream os("temp_0.bmp", std::ios::binary);
	unsigned char signature[2] = { 'B', 'M' };
	unsigned int fileSize = 14 + 40 + new_width * new_height * 3;
	unsigned int reserved = 0;
	unsigned int offset = 14 + 40;
	unsigned int headerSize = 40;
	unsigned int dimensions1 = new_width;
	unsigned int dimensions2 = new_height;
	unsigned short colorPlanes = 1;
	unsigned short bpp = 24;
	unsigned int compression = 0;
	unsigned int imgSize = new_width * new_height * 3;
	unsigned int resolution[2] = { 2795, 2795 };
	unsigned int pltColors = 0;
	unsigned int impColors = 0;
	os.write(reinterpret_cast<char*>(signature), sizeof(signature));
	os.write(reinterpret_cast<char*>(&fileSize), sizeof(fileSize));
	os.write(reinterpret_cast<char*>(&reserved), sizeof(reserved));
	os.write(reinterpret_cast<char*>(&offset), sizeof(offset));
	os.write(reinterpret_cast<char*>(&headerSize), sizeof(headerSize));
	os.write(reinterpret_cast<char*>(&dimensions1), sizeof(dimensions1));
	os.write(reinterpret_cast<char*>(&dimensions2), sizeof(dimensions2));
	os.write(reinterpret_cast<char*>(&colorPlanes), sizeof(colorPlanes));
	os.write(reinterpret_cast<char*>(&bpp), sizeof(bpp));
	os.write(reinterpret_cast<char*>(&compression), sizeof(compression));
	os.write(reinterpret_cast<char*>(&imgSize), sizeof(imgSize));
	os.write(reinterpret_cast<char*>(resolution), sizeof(resolution));
	os.write(reinterpret_cast<char*>(&pltColors), sizeof(pltColors));
	os.write(reinterpret_cast<char*>(&impColors), sizeof(impColors));

	px_arr old, young_w, young_wh;
	young_w.r = new unsigned int[height * new_width];
	young_w.g = new unsigned int[height * new_width];
	young_w.b = new unsigned int[height * new_width];
	young_wh.r = new unsigned int[new_height * new_width];
	young_wh.g = new unsigned int[new_height * new_width];
	young_wh.b = new unsigned int[new_height * new_width];
	old = reading(width, height, file);
	small_width(old.r, old.g, old.b, width, height, new_width, young_w.r, young_w.g, young_w.b);

	small_height(young_w.r, young_w.g, young_w.b, new_width, height, new_height, young_wh.r, young_wh.g, young_wh.b);
	writing(young_wh, new_height, os, new_width);

	os.close();

	return 0;
}